#include <stdint.h>
#include <string.h>
#include <stdio.h>
#include <assert.h>
#include "plchain.h"
#include "debug.h"
#include <time.h>
#include "plkernels.cu"

hostMemPtr host_mem_ptrs[NUM_STREAMS];
deviceMemPtr device_mem_ptrs[NUM_STREAMS];
// hipStream_t streams[NUM_STREAMS]; // init streams
// hipEvent_t events[NUM_STREAMS];

// i = __sync_fetch_and_add(i, delta); // NOTE: atomic add, i starts from thread_idx
int total_frags = 0;
int total_tasks = 0;
int max_awaiting_tasks = 0;
int awaiting_tasks = 0;
pthread_mutex_t pltask_lock; // lock for task append
pthread_cond_t pltask_cv;

size_t total_n = 0; // NOTE: index where the array is stored 
int size = 0;
int grid_dim = 0;
int cut_num = 0;

int64_t *get_p(int64_t *p, int64_t n, size_t index) {
    uint16_t *rel = host_mem_ptrs[0].p+index;
    if (!p) int64_t* p = (int64_t*)malloc(sizeof(int64_t)*n);
    for (int i = 0; i < n; ++i) {
        if (rel[i] == 0)
            p[i] = -1;
        else
            p[i] = i - rel[i];
    }
    return p; // TODO: make sure p is freed after using it
}

int32_t *get_f(int64_t n, size_t index) {
    return host_mem_ptrs[0].f+index;
}

int pltask_init(int num_threads, int num_seqs) { 

    size = num_threads;
    max_awaiting_tasks = num_threads;
    total_frags = num_seqs;
    pthread_mutex_init(&pltask_lock, 0);
	pthread_cond_init(&pltask_cv, 0);
    // NOTE: allocate pin memory for each stream
    size_t avail_mem_stream = MEM_GPU;
    avail_mem_stream = MEM_GPU/NUM_STREAMS * 1e9; // split memory for each stream
    // memory per anchor = ax + ay + range + f + p + (start_idx + read_end_idx + cut_start_idx + cut)
    // size: F1 = ax + ay + range + f + p; F2 = start_idx + read_end_idx + cut_start_idx; F3 = cut
    int64_t F1 = 8+8+4+4+2, F2 = 8+8+8, F3 = 8;
    int64_t P1 = ANCHORS_PER_BLOCK, P2 = ANCHORS_PER_CUT;
    // avail_memory = (F1 + F2/ANCHORS_PER_BLOCK + F3/ANCHORS_PER_CUT) * num_anchors
    int64_t max_anchors_stream = (avail_mem_stream*P1*P2) / (F1*P1*P2 + F2*P2 + F3*P1); // ignore misc as anchors cannot just fit whole memory
    int64_t max_grid = max_anchors_stream / ANCHORS_PER_BLOCK;
    int64_t max_num_cut = max_anchors_stream / ANCHORS_PER_CUT;

    for (int i = 0; i < NUM_STREAMS; ++i) {
        // hipStreamCreate(&streams[i]);
        // hipEventCreate(&events[i]);
        cudaCheck();
        // set up host memory pointers
        host_mem_ptrs[i].index = -1; // -1 means unused stream
        hipHostMalloc((void**)&host_mem_ptrs[i].ax, max_anchors_stream * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].ay, max_anchors_stream * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].f, max_anchors_stream * sizeof(int32_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].p, max_anchors_stream * sizeof(uint16_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].start_idx, max_grid * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].read_end_idx, max_grid * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].cut_start_idx, max_grid * sizeof(size_t));
        // set up GPU memory pointers
        hipMalloc(&device_mem_ptrs[i].d_ax, max_anchors_stream * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_ay, max_anchors_stream * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_range, max_anchors_stream * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_f, max_anchors_stream * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_p, max_anchors_stream * sizeof(uint16_t));    
        hipMalloc(&device_mem_ptrs[i].d_cut, max_num_cut * sizeof(size_t));
        hipMalloc(&device_mem_ptrs[i].d_start_idx, sizeof(size_t) * max_grid);
        hipMalloc(&device_mem_ptrs[i].d_read_end_idx, sizeof(size_t) * max_grid);
        hipMalloc(&device_mem_ptrs[i].d_cut_start_idx, sizeof(size_t) * max_grid);
        cudaCheck();
    }
    return 0;
}

size_t pltask_append(int64_t n, mm128_t *a, int max_dist_x, int max_dist_y, int bw, int max_skip, int max_iter,
    float chn_pen_gap, float chn_pen_skip, int is_cdna, int n_seg) {  
    // NOTE: This function must be called inside a critical section

    pthread_mutex_lock(&pltask_lock);
    awaiting_tasks++; 
    int task_id = total_tasks++;

    // record how many anchors appended
    size_t key = total_n;
    size_t idx = total_n;
    total_n += n;

    if (max_dist_x < bw) max_dist_x = bw;
	if (max_dist_y < bw && !is_cdna) max_dist_y = bw;

    // copy data to pinned memory
    hostMemPtr *host_mem_ptr = host_mem_ptrs + 0;
    deviceMemPtr *device_mem_ptr = device_mem_ptrs + 0;

    int block_num = (n - 1) / MAX_ANCHOR_PER_BLOCK + 1;

    host_mem_ptr->start_idx[grid_dim] = idx;
    size_t end_idx = idx + MAX_ANCHOR_PER_BLOCK;
    host_mem_ptr->read_end_idx[grid_dim] = idx + n;
    host_mem_ptr->cut_start_idx[grid_dim] = cut_num;
    for (int j = 1; j < block_num; j++) {
        cut_num += MAX_IT_BLOCK_RANGE;
        host_mem_ptr->start_idx[grid_dim + j] = end_idx;
        end_idx = host_mem_ptr->start_idx[grid_dim + j] + MAX_ANCHOR_PER_BLOCK;
        host_mem_ptr->read_end_idx[grid_dim + j] = idx + n;
        host_mem_ptr->cut_start_idx[grid_dim + j] = cut_num;
    }

    cut_num += (n - (block_num - 1) * MAX_ANCHOR_PER_BLOCK - 1) / NUM_THREADS_RANGE + 1;

    grid_dim += block_num;

    // copy anchors to pin memory
    for (int j = 0; j < n; j++){
        host_mem_ptr->ax[idx] = a[j].x;
        host_mem_ptr->ay[idx] = a[j].y; 
        ++idx;
    }

    if (awaiting_tasks == max_awaiting_tasks || total_tasks == total_frags) { // TODO: when it is not a multiple of n_threads
        fprintf(stderr, "[M: %s] Launch chaining kernel with %d seqs, %d / %d\n", __func__, awaiting_tasks, total_tasks, total_frags);
        Misc misc_info;
        misc_info.bw = bw;
        misc_info.max_skip = max_skip;
        misc_info.max_iter = max_iter;
        misc_info.max_dist_x = max_dist_x;
        misc_info.max_dist_y = max_dist_y;
        misc_info.is_cdna = is_cdna;
        misc_info.chn_pen_gap = chn_pen_gap;
        misc_info.chn_pen_skip = chn_pen_skip;
        misc_info.n_seg = n_seg;
		pltask_launch(&misc_info);
        awaiting_tasks = 0;
        pthread_cond_broadcast(&pltask_cv);
    } else {
        pthread_cond_wait(&pltask_cv, &pltask_lock);
    }
    pthread_mutex_unlock(&pltask_lock);

    fprintf(stderr, "[M: %s] ready to continue, %d / %d\n", __func__, task_id, total_frags);
    return key;
}

int pltask_launch(Misc *misc_info) {

    hostMemPtr *host_mem_ptr = host_mem_ptrs;
    deviceMemPtr *device_mem_ptr = device_mem_ptrs;
    device_mem_ptr->num_cut = cut_num;

    dim3 DimBlock0(NUM_THREADS_RANGE, 1, 1);
    dim3 DimGrid0(grid_dim, 1, 1);

    hipMemcpy(device_mem_ptr->d_ax, host_mem_ptr->ax, sizeof(int64_t)*total_n, hipMemcpyHostToDevice);
    hipMemcpy(device_mem_ptr->d_ay, host_mem_ptr->ay, sizeof(int64_t)*total_n, hipMemcpyHostToDevice);
    hipMemcpy(device_mem_ptr->d_start_idx, host_mem_ptr->start_idx, sizeof(size_t)*grid_dim, hipMemcpyHostToDevice);
    hipMemcpy(device_mem_ptr->d_read_end_idx, host_mem_ptr->read_end_idx, sizeof(size_t)*grid_dim, hipMemcpyHostToDevice);
    
    hipMemcpy(device_mem_ptr->d_cut_start_idx, host_mem_ptr->cut_start_idx, sizeof(size_t)*grid_dim, hipMemcpyHostToDevice);
    hipMemset(device_mem_ptr->d_cut, 0xff, sizeof(size_t)*cut_num);
    cudaCheck();

    fprintf(stderr, "[M: %s] Launch range selection\n", __func__);
    range_selection_kernel_naive<<<DimGrid0, DimBlock0>>>(device_mem_ptr->d_ax, device_mem_ptr->d_start_idx, device_mem_ptr->d_read_end_idx, 
                                                                device_mem_ptr->d_range, device_mem_ptr->d_cut, device_mem_ptr->d_cut_start_idx);
    cudaCheck();

    upload_misc(misc_info);

    cudaCheck();
    int griddim = (cut_num-1)/NUM_SEG_PER_BLOCK + 1;
    dim3 DimBlock1(NUM_THREADS_SCORE, 1, 1);
    dim3 DimGrid1(griddim, 1, 1);
    fprintf(stderr, "[M: %s] Launch score generation\n", __func__);
    score_generation_naive<<<DimGrid1, DimBlock1>>>(device_mem_ptr->d_ax, device_mem_ptr->d_ay, device_mem_ptr->d_range, 
                                            device_mem_ptr->d_cut, device_mem_ptr->d_f, device_mem_ptr->d_p, total_n, cut_num);
    cudaCheck();

    // copy f and p back to host
    hipMemcpy(host_mem_ptr->f, device_mem_ptr->d_f, sizeof(int32_t)*total_n, hipMemcpyDeviceToHost);
    hipMemcpy(host_mem_ptr->p, device_mem_ptr->d_p, sizeof(uint16_t)*total_n, hipMemcpyDeviceToHost);

    // cuda sync device
    
    // NOTE: reset the grid size 
    grid_dim = 0;
    cut_num = 0;
    
    return 0;
}


