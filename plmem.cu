#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>
#include <stdio.h>
#include <assert.h>
#include <time.h>

#include "plchain.h"
#include "plthread.h"
#include "debug.h"
#include "plkernel.cu"

static task_t *chaining_tasks;
static task_t *alignment_tasks;
static size_t chain_count; // +=n_a for every chaining task
static size_t align_count; // +=n_a for every alignment task
static long chain_index, done_chain_index;
static long align_index, done_align_index;
static int64_t max_anchors_stream;
static int64_t max_grid;
static int64_t max_num_cut;
static int task_count; // count how many tasks processed 

hostMemPtr host_mem_ptrs[NUM_STREAMS];
deviceMemPtr device_mem_ptrs[NUM_STREAMS];
hipStream_t streams[NUM_STREAMS]; // init streams
hipEvent_t events[NUM_STREAMS];

static bool gpu_busy;
pthread_mutex_t pltask_lock; // lock for task append
pthread_cond_t pltask_cv;

// TODO: put this into plscore?
// __constant__ Misc misc;

void set_task_misc(task_t *task, int max_dist_x, int max_dist_y, const mm_mapopt_t *opt,
    float chn_pen_gap, float chn_pen_skip, int is_cdna, int n_seg) {
    task->misc.bw = opt->bw;
    task->misc.max_skip = opt->max_chain_skip;
    task->misc.max_iter = opt->max_chain_iter;
    task->misc.min_cnt = opt->min_cnt;
    task->misc.min_sc = opt->min_chain_score;
    task->misc.max_dist_x = max_dist_x;
    task->misc.max_dist_y = max_dist_y;
    task->misc.is_cdna = is_cdna;
    task->misc.chn_pen_gap = chn_pen_gap;
    task->misc.chn_pen_skip = chn_pen_skip;
    task->misc.n_seg = n_seg;
}

// TODO: probably sill need a lock as atomic add has no boundary check

int pltask_init(int num_seqs) {
    gpu_busy = false;
    chain_count = align_count = 0;
    chain_index = align_index = done_chain_index = done_align_index = 0;
    pthread_mutex_init(&pltask_lock, 0);
	pthread_cond_init(&pltask_cv, 0);
    chaining_tasks = (task_t *) malloc(sizeof(task_t)*num_seqs);
    alignment_tasks = (task_t *) malloc(sizeof(task_t)*num_seqs);
    task_count = 0;

    // NOTE: allocate pin memory for each stream
    size_t avail_mem_stream = MEM_GPU;
    avail_mem_stream = MEM_GPU/NUM_STREAMS * 1e9; // split memory for each stream
    // memory per anchor = ax + ay + range + f + p + (start_idx + read_end_idx + cut_start_idx + cut)
    // size: F1 = ax + ay + range + f + p; F2 = start_idx + read_end_idx + cut_start_idx; F3 = cut
    int64_t F1 = 8+8+4+4+2, F2 = 8+8+8, F3 = 8;
    int64_t P1 = ANCHORS_PER_BLOCK, P2 = ANCHORS_PER_CUT;
    // avail_memory = (F1 + F2/ANCHORS_PER_BLOCK + F3/ANCHORS_PER_CUT) * num_anchors
    max_anchors_stream = (avail_mem_stream*P1*P2) / (F1*P1*P2 + F2*P2 + F3*P1); // ignore misc as anchors cannot just fit whole memory
    max_grid = max_anchors_stream / ANCHORS_PER_BLOCK;
    max_num_cut = max_anchors_stream / ANCHORS_PER_CUT;

    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&events[i]);
        cudaCheck();
        // set up host memory pointers
        host_mem_ptrs[i].index = -1; // -1 means unused stream
        hipHostMalloc((void**)&host_mem_ptrs[i].ax, max_anchors_stream * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].ay, max_anchors_stream * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].f, max_anchors_stream * sizeof(int32_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].p, max_anchors_stream * sizeof(uint16_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].start_idx, max_grid * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].read_end_idx, max_grid * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].cut_start_idx, max_grid * sizeof(size_t));
        // set up GPU memory pointers
        hipMalloc(&device_mem_ptrs[i].d_ax, max_anchors_stream * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_ay, max_anchors_stream * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_range, max_anchors_stream * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_f, max_anchors_stream * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_p, max_anchors_stream * sizeof(uint16_t));    
        hipMalloc(&device_mem_ptrs[i].d_cut, max_num_cut * sizeof(size_t));
        hipMalloc(&device_mem_ptrs[i].d_start_idx, sizeof(size_t) * max_grid);
        hipMalloc(&device_mem_ptrs[i].d_read_end_idx, sizeof(size_t) * max_grid);
        hipMalloc(&device_mem_ptrs[i].d_cut_start_idx, sizeof(size_t) * max_grid);
        cudaCheck();
    }
    return 0;
}

// NOTE: i is qry sequence offset
const task_t *pltask_chain_get(long i) {
    return chaining_tasks+i;
}

const task_t *pltask_align_get(long i) {
    return alignment_tasks+i;
}

/*********************** Thread function calls start ************************/

// FIXME: because memory reuse is difficult for multi-stream, start from post buffer

int plchain_append(int max_dist_x, int max_dist_y, const mm_mapopt_t *opt,
    float chn_pen_gap, float chn_pen_skip, int is_cdna,
    int n_seg, int64_t n,  // NOTE: n is number of anchors
    mm128_t *a,            // NOTE: a is ptr to anchors.
    void *km, int n_mini_pos, uint64_t *mini_pos, uint32_t hash, long i) {  
    // TODO: increment the count, check if memory is full
    pthread_mutex_lock(&pltask_lock);
    
    auto *task = chaining_tasks + chain_index++;
    if (task->status != EMPTY) {
        // wrong slot
        // FIXME: how to deal with this?
        return -1;
    }
    
    task->i = i;
    task->a = a;
    task->hash = hash;
    task->mini_pos = mini_pos;
    task->n_mini_pos = n_mini_pos;
    task->size = n;
    task->n_regs0 = 0;
    set_task_misc(task, max_dist_x, max_dist_y, opt, chn_pen_gap, chn_pen_skip, is_cdna, n_seg);

    // check if reach buffer capacity 
    if (chain_count + n > max_anchors_stream) {
        // TODO: call gpu function to copy memory to pin memory and launch stream
        // update done_index when GPU finished
        int ret = plchain_stream_launch(chain_index-1); // current sequence exceed memory
        done_chain_index = chain_index; 
        chain_count = 0;
    }

    task->offset = chain_count;
    chain_count += n;

    pthread_mutex_unlock(&pltask_lock);

    // NOTE: memory copy happens in gpu to avoid two global var increment which require lock

    return 0;
}

int plchain_check(long i) {

    pthread_mutex_lock(&pltask_lock);

    Status status = chaining_tasks[i].status;

    if (status == IDLE)  {

    } else if (status == TASK_ON) {

    } else if (status == TASK_END) {

    } else {
        
    }



    switch (status)
    {
    case IDLE:
        // TODO: start gpu if gpu_busy is false
        break;

    case TASK_ON:
        // TODO: check all streams
        // TODO: set status of task
        // TODO: backtracking 
        break;

    case TASK_END:
        // TODO: misc after gpu is done
        // TODO: backtracking 
        break;
    default:
        break;
    }

    pthread_mutex_unlock(&pltask_lock);

    return 0; 
    // TODO: return 0 if gpu is done,
    // return -1 if gpu just starts
    // return 1 if gpu has start but not finished yet  
}

// TODO: alignment tasks append and check

/*********************** Thread function calls end ************************/

/*********************** GPU loops start ************************/

int plchain_stream_launch(long end_chain_index) {
    // find an available stream
    int stream_idx = task_count;
    if (task_count >= NUM_STREAMS) {
        stream_idx = -1;
        while (stream_idx == -1) {
            for (int t = 0; t < NUM_STREAMS; ++t) {
                if (!hipEventQuery(events[t])) {
                    stream_idx = t;
                    // FIXME: unnecessary recreate?
                    hipEventDestroy(events[stream_idx]);
                    hipEventCreate(&events[stream_idx]);
                    break;
                }
            }
        }
        // TODO: collect f and p of last stream, set status 
        // TODO: hostMemPtr should contain index of tasks 
        hostMemPtr *host_mem_ptr = host_mem_ptrs+stream_idx;
        int64_t offset = 0;
        for (int i = host_mem_ptr->index; i < host_mem_ptr->index+host_mem_ptr->size; ++i) {
            auto *task = chaining_tasks + i;
            task->status = TASK_END;
            task->f = (int32_t *) malloc(sizeof(int32_t) * task->size);
            task->p = (int64_t *) malloc(sizeof(int64_t) * task->size);
            memcpy(task->f, host_mem_ptr->f + offset, sizeof(int32_t) * task->size);
            memcpy(task->p, host_mem_ptr->p + offset, sizeof(int64_t) * task->size);
            offset += task->size;
        }
        // call backtracking? but has no multithread, we still need in GPU backtracking 
        // TODO: afterwards tasks

    } 
    // launch new task
    hostMemPtr *host_mem_ptr = host_mem_ptrs+stream_idx;
    deviceMemPtr *device_mem_ptr = device_mem_ptrs+stream_idx;
    hipStream_t *stream = streams+stream_idx;
    hipEvent_t *event = events+stream_idx;
    int size = (int) (end_chain_index - done_chain_index);
    size_t total_n = chain_count;
    size_t griddim = 0;
    size_t idx = 0;
    size_t cut_num = 0;
    host_mem_ptr->index = (int) done_chain_index;
    host_mem_ptr->size = size;

    device_mem_ptr->total_n = total_n;

    for (long i = done_chain_index; i < end_chain_index; ++i) {
        auto *task = chaining_tasks + i;
        int n = task->size;
        int block_num = (n - 1) / MAX_ANCHOR_PER_BLOCK + 1;

        host_mem_ptr->start_idx[griddim] = idx;
        size_t end_idx = idx + MAX_ANCHOR_PER_BLOCK;
        host_mem_ptr->read_end_idx[griddim] = idx + n;
        host_mem_ptr->cut_start_idx[griddim] = cut_num;
        for (int j=1; j<block_num; j++){
            cut_num += MAX_IT_BLOCK_RANGE;
            host_mem_ptr->start_idx[griddim + j] = end_idx;
            end_idx = host_mem_ptr->start_idx[griddim + j] + MAX_ANCHOR_PER_BLOCK;
            host_mem_ptr->read_end_idx[griddim + j] = idx + n;
            host_mem_ptr->cut_start_idx[griddim + j] = cut_num;
        }
        cut_num += (n - (block_num -1) * MAX_ANCHOR_PER_BLOCK - 1) / NUM_THREADS_RANGE + 1;
        end_idx = idx + n;

        griddim += block_num;

        // copy anchors to pin memory
        for (int j =0; j < n; j++){
            host_mem_ptr->ax[idx] = task->a[j].x;
            host_mem_ptr->ay[idx] = task->a[j].y; 
            ++idx;
        }
        // free(task->a); // TODO: check if OK to free here
    }
    device_mem_ptr->num_cut = cut_num;

    dim3 DimBlock0(NUM_THREADS_RANGE, 1, 1);
    dim3 DimGrid0(griddim,1,1);

    hipMemcpyAsync(device_mem_ptr->d_ax, host_mem_ptr->ax, sizeof(int64_t)*total_n, hipMemcpyHostToDevice, *stream);
    hipMemcpyAsync(device_mem_ptr->d_ay, host_mem_ptr->ay, sizeof(int64_t)*total_n, hipMemcpyHostToDevice, *stream);
    hipMemcpyAsync(device_mem_ptr->d_start_idx, host_mem_ptr->start_idx, sizeof(size_t)*griddim, hipMemcpyHostToDevice, *stream);
    hipMemcpyAsync(device_mem_ptr->d_read_end_idx, host_mem_ptr->read_end_idx, sizeof(size_t)*griddim, hipMemcpyHostToDevice, *stream);
    
    hipMemcpyAsync(device_mem_ptr->d_cut_start_idx, host_mem_ptr->cut_start_idx, sizeof(size_t)*griddim, hipMemcpyHostToDevice, *stream);
    hipMemsetAsync(device_mem_ptr->d_cut, 0xff, sizeof(size_t)*cut_num, *stream);
    cudaCheck();

    range_selection_kernel_naive<<<DimGrid0, DimBlock0, 0, *stream>>>(device_mem_ptr->d_ax, device_mem_ptr->d_start_idx, device_mem_ptr->d_read_end_idx, 
                                                                device_mem_ptr->d_range, device_mem_ptr->d_cut, device_mem_ptr->d_cut_start_idx);
    cudaCheck();

    upload_misc(stream_idx, &chaining_tasks[0].misc, stream);

    cudaCheck();
    griddim = (cut_num-1)/NUM_SEG_PER_BLOCK + 1;
    dim3 DimBlock1(NUM_THREADS_SCORE, 1, 1);
    dim3 DimGrid1(griddim, 1, 1);
    score_generation_naive<<<DimGrid1, DimBlock1, 0, *stream>>>(device_mem_ptr->d_ax, device_mem_ptr->d_ay, device_mem_ptr->d_range, 
                                            device_mem_ptr->d_cut, device_mem_ptr->d_f, device_mem_ptr->d_p, total_n, cut_num);
    cudaCheck();

    // copy f and p back to host
    hipMemcpyAsync(host_mem_ptr->f, device_mem_ptr->d_f, sizeof(int32_t)*total_n, hipMemcpyDeviceToHost, *stream);
    hipMemcpyAsync(host_mem_ptr->p, device_mem_ptr->d_p, sizeof(uint16_t)*total_n, hipMemcpyDeviceToHost, *stream);
    // TODO: try implement backtracking on GPU

    cudaCheck();

    hipEventRecord(*event, *stream);
    cudaCheck();

    // TODO: do alignment here for correctness test

    return 0;
}




double dynamic_stream_chain_loop(input_iter* input_arr, int total_reads) {
    // NOTE: return duration of this cpu batch
    // NUM_STREAMS must be more than one
    assert(NUM_STREAMS > 1);
    hostMemPtr host_mem_ptrs[NUM_STREAMS];
    deviceMemPtr device_mem_ptrs[NUM_STREAMS];
    double dura = 0.0;
    clock_t clk_start, clk_end;
    clk_start = time(NULL);
    // clk_start = clock();

    // set up stream and memory pointers
    hipStream_t * streams = new hipStream_t[NUM_STREAMS]; // init streams
    hipEvent_t * events = new hipEvent_t[NUM_STREAMS];
    // FIXME: is it possible for a read to be too long to fit in one stream?
    size_t avail_mem_stream = MEM_GPU;
    avail_mem_stream = MEM_GPU/NUM_STREAMS * 1e9; // split memory for each stream
#ifdef DEBUG_CHECK
    fprintf(stderr, "[M: %s] memory per stream: %zuB\n", __func__, avail_mem_stream);    
#endif // DEBUG_CHECK
    // memory per anchor = ax + ay + range + f + p + (start_idx + read_end_idx + cut_start_idx + cut)
    // size: F1 = ax + ay + range + f + p; F2 = start_idx + read_end_idx + cut_start_idx; F3 = cut
    int64_t F1 = 8+8+4+4+2, F2 = 8+8+8, F3 = 8;
    int64_t P1 = ANCHORS_PER_BLOCK, P2 = ANCHORS_PER_CUT;
    // avail_memory = (F1 + F2/ANCHORS_PER_BLOCK + F3/ANCHORS_PER_CUT) * num_anchors
    int64_t max_anchors_stream = (avail_mem_stream*P1*P2) / (F1*P1*P2 + F2*P2 + F3*P1); // ignore misc as anchors cannot just fit whole memory
    int64_t max_grid = max_anchors_stream / ANCHORS_PER_BLOCK;
    int64_t max_num_cut = max_anchors_stream / ANCHORS_PER_CUT;
#ifdef DEBUG_CHECK
    fprintf(stderr, "[M: %s] per stream max_anchors: %ld, max_grid: %ld, max_cut: %ld\n", __func__, max_anchors_stream, max_grid, max_num_cut);    
#endif // DEBUG_CHECK

    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&events[i]);
        cudaCheck();
        // set up host memory pointers
        host_mem_ptrs[i].index = -1; // -1 means unused stream
        hipHostMalloc((void**)&host_mem_ptrs[i].ax, max_anchors_stream * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].ay, max_anchors_stream * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].f, max_anchors_stream * sizeof(int32_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].p, max_anchors_stream * sizeof(uint16_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].start_idx, max_grid * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].read_end_idx, max_grid * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].cut_start_idx, max_grid * sizeof(size_t));
        // set up GPU memory pointers
        hipMalloc(&device_mem_ptrs[i].d_ax, max_anchors_stream * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_ay, max_anchors_stream * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_range, max_anchors_stream * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_f, max_anchors_stream * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_p, max_anchors_stream * sizeof(uint16_t));    
        hipMalloc(&device_mem_ptrs[i].d_cut, max_num_cut * sizeof(size_t));
        hipMalloc(&device_mem_ptrs[i].d_start_idx, sizeof(size_t) * max_grid);
        hipMalloc(&device_mem_ptrs[i].d_read_end_idx, sizeof(size_t) * max_grid);
        hipMalloc(&device_mem_ptrs[i].d_cut_start_idx, sizeof(size_t) * max_grid);
        cudaCheck();
    }

    // schedule streams until all reads finished 
    int processed_reads = 0; // reads processed so far
    input_iter* curr_arr = input_arr;
    int batch_count = 0; // batchs launched so far
    while (processed_reads < total_reads) {
        int read_idx;
        int64_t num_anchors = 0;
        // assign anchors for stream
        for (read_idx = processed_reads; read_idx < total_reads; ++read_idx) {
            if (num_anchors + input_arr[read_idx].n > max_anchors_stream) {
            #ifdef DEBUG_CHECK
                fprintf(stderr, "[M: %s] stream_mem fills up after %d reads and %ld anchors\n", __func__, read_idx - processed_reads, num_anchors);    
            #endif // DEBUG_CHECK
                break;
            }
            num_anchors += input_arr[read_idx].n; // increment only when memory is available
        }

        int stream_idx = batch_count;
        // find usable stream
        if (batch_count >= NUM_STREAMS) {
            stream_idx = -1;
            while (stream_idx == -1) {
                for (int t = 0; t < NUM_STREAMS; ++t) {
                    if (!hipEventQuery(events[t])) {
                        stream_idx = t;
                        // FIXME: unnecessary recreate?
                        hipEventDestroy(events[stream_idx]);
                        hipEventCreate(&events[stream_idx]);
                        break;
                    }
                }
            }
            cudaCheck();
#ifdef DEBUG_CHECK
            int sync_iter = stream_idx;
            // TODO: check correctness
            int index = host_mem_ptrs[sync_iter].index; // index here is read index
            // fprintf(stderr, "[M::%s] correctness check index: %d\n", __func__, index);
            int64_t read_start = 0;
            input_iter *prev_arr = input_arr + index;
            // range check
            if (is_debug_range() != -1) {
                int32_t* range = (int32_t*)malloc(sizeof(int32_t) * device_mem_ptrs[sync_iter].total_n);
                hipMemcpy(range, device_mem_ptrs[sync_iter].d_range, sizeof(int32_t)*device_mem_ptrs[sync_iter].total_n, hipMemcpyDeviceToHost);
                // fprintf(stderr, "[M::%s] Start range check\n", __func__);
                for (int i = 0; i < host_mem_ptrs[sync_iter].size; i++){
                    debug_print_successor_range(range+read_start, prev_arr[i].n);
                    debug_check_range(range + read_start, prev_arr[i].range, prev_arr[i].n);
                    read_start += prev_arr[i].n;
                }
                free(range);
            }
            // score check
            fprintf(stderr, "[M::%s] Start score check of %d reads\n", __func__, host_mem_ptrs[sync_iter].size);
            read_start = 0;
            for (int i = 0; i < host_mem_ptrs[sync_iter].size; i++) {
                // int64_t* p = (int64_t*)malloc(sizeof(int64_t)*prev_arr[i].n);
                // p_rel2idx(host_mem_ptrs[sync_iter].p + read_start, p, prev_arr[i].n);
                // debug_check_score(p, host_mem_ptrs[sync_iter].f + read_start, prev_arr[i].p, prev_arr[i].f, prev_arr[i].n);
                debug_check_score_auto_trans(host_mem_ptrs[sync_iter].p + read_start, host_mem_ptrs[sync_iter].f + read_start, prev_arr[i].p, prev_arr[i].f, prev_arr[i].n);
                read_start += prev_arr[i].n;
            }
            fprintf(stderr, "[M::%s] End score check\n", __func__);
#endif
            // TODO: read p, f
            // host_mem_ptrs[sync_iter].f
            // host_mem_ptrs[sync_iter].p
        }
        batch_count++;
        int batch_size = read_idx - processed_reads;
        host_mem_ptrs[stream_idx].index = processed_reads; // record read index
        host_mem_ptrs[stream_idx].size = batch_size; // record batch size (num of reads processed)
        device_mem_ptrs[stream_idx].total_n = num_anchors;
        fprintf(stderr, "[M::%s] Start stream with %d reads\n", __func__, batch_size);
        stream_range_selection(curr_arr, host_mem_ptrs + stream_idx, device_mem_ptrs + stream_idx, batch_size, (void *)(&streams[stream_idx]));
        cudaCheck();
        stream_score_generation(curr_arr, host_mem_ptrs + stream_idx, device_mem_ptrs + stream_idx, batch_size, (void *)(&streams[stream_idx]), (void *)(&events[stream_idx]));
        cudaCheck();
        curr_arr += batch_size;
        processed_reads = read_idx;
#ifdef DEBUG_CHECK
        fprintf(stderr, "[M: %s] -----reads in progress %d/%d----- \n", __func__, processed_reads, total_reads);    
#endif // DEBUG_CHECK
    }

    fprintf(stderr, "[M::%s] Sync up all streams\n", __func__);
    // sync all the streams
    for (int sync_iter = 0; sync_iter < NUM_STREAMS; ++sync_iter) {
        int index = host_mem_ptrs[sync_iter].index;
        // fprintf(stderr, "[M::%s] final sync all index: %d, sync_iter: %d\n", __func__, index, sync_iter);
        if (index != -1) {
            hipStreamSynchronize(streams[sync_iter]);
            cudaCheck();
            // TODO: check correctness
#ifdef DEBUG_CHECK
            // fprintf(stderr, "[M::%s] correctness check index: %d, batch_size: %d\n", __func__, index, batch_size);
            int64_t read_start = 0;
            input_iter *prev_arr = input_arr + index;
            // range check
            if (is_debug_range() != -1) {
                int32_t* range = (int32_t*)malloc(sizeof(int32_t) * device_mem_ptrs[sync_iter].total_n);
                hipMemcpy(range, device_mem_ptrs[sync_iter].d_range, sizeof(int32_t)*device_mem_ptrs[sync_iter].total_n, hipMemcpyDeviceToHost);
                // fprintf(stderr, "[M::%s] Start range check\n", __func__);
                for (int i = 0; i < host_mem_ptrs[sync_iter].size; i++){
                    debug_print_successor_range(range+read_start, prev_arr[i].n);
                    debug_check_range(range + read_start, prev_arr[i].range, prev_arr[i].n);
                    read_start += prev_arr[i].n;
                }
                free(range);
            }
            // score check
            read_start = 0;
            fprintf(stderr, "[M::%s] Start score check of %d reads\n", __func__, host_mem_ptrs[sync_iter].size);
            for (int i = 0; i < host_mem_ptrs[sync_iter].size; i++) {
                // NOTE: this temp buffer needs to be considered in memory alloc
                // int64_t* p = (int64_t*)malloc(sizeof(int64_t)*prev_arr[i].n);
                // p_rel2idx(host_mem_ptrs[sync_iter].p + read_start, p, prev_arr[i].n);
                // debug_check_score(p, host_mem_ptrs[sync_iter].f + read_start, prev_arr[i].p, prev_arr[i].f, prev_arr[i].n);
                debug_check_score_auto_trans(host_mem_ptrs[sync_iter].p + read_start, host_mem_ptrs[sync_iter].f + read_start, prev_arr[i].p, prev_arr[i].f, prev_arr[i].n);
                read_start += prev_arr[i].n;
            }
            fprintf(stderr, "[M::%s] End score check\n", __func__);
#endif
        } else break;
    }

    // free all memory
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(events[i]);
        cudaCheck();

        // free host memory
        hipHostFree(host_mem_ptrs[i].ax);
        hipHostFree(host_mem_ptrs[i].ay);
        hipHostFree(host_mem_ptrs[i].f);
        hipHostFree(host_mem_ptrs[i].p);
        hipHostFree(host_mem_ptrs[i].start_idx);
        hipHostFree(host_mem_ptrs[i].read_end_idx);
        hipHostFree(host_mem_ptrs[i].cut_start_idx);
        // free device memory
        hipFree(device_mem_ptrs[i].d_ax);
        hipFree(device_mem_ptrs[i].d_ay);
        hipFree(device_mem_ptrs[i].d_range);
        hipFree(device_mem_ptrs[i].d_f);
        hipFree(device_mem_ptrs[i].d_p);
        hipFree(device_mem_ptrs[i].d_cut);
        hipFree(device_mem_ptrs[i].d_start_idx);
        hipFree(device_mem_ptrs[i].d_read_end_idx);
        hipFree(device_mem_ptrs[i].d_cut_start_idx);
        cudaCheck();
    }
    delete[] streams;
    delete[] events;

    clk_end = time(NULL);
    // clk_end = clock();
    dura =  (double) (clk_end - clk_start);
    fprintf(stderr, "[Dynamic stream chaining] ======CPU Batch Chaining Run Time: %lf secs\n", dura);
    // printf("[Stream chaining] ======Chaining Run Time: %lf secs\n", ((double) (clk_end - clk_start)) / CLOCKS_PER_SEC);
    return dura;

}

void stream_chain_loop(input_iter* input_arr, int total_iter) {
    // NUM_STREAMS must be more than one
    assert(NUM_STREAMS > 1);
    hostMemPtr host_mem_ptrs[NUM_STREAMS];
    deviceMemPtr device_mem_ptrs[NUM_STREAMS];
    double range_selection_dura = 0.0, score_generation_dura = 0.0, cpu_chain_dura = 0.0;
    clock_t clk_start, clk_end;
    clk_start = time(NULL);
    // clk_start = clock();
    
    // set up stream and memory pointers
    hipStream_t * streams = new hipStream_t[NUM_STREAMS]; // init streams
    hipEvent_t * events = new hipEvent_t[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
        hipEventCreate(&events[i]);
        cudaCheck();

        // NOTE: ax ay range require dynamic size, others have max size limitation
        // set up host memory pointers
        host_mem_ptrs[i].index = -1;
        hipHostMalloc((void**)&host_mem_ptrs[i].ax, MAX_NUM_ANCHORS * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].ay, MAX_NUM_ANCHORS * sizeof(int64_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].f, MAX_NUM_ANCHORS * sizeof(int32_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].p, MAX_NUM_ANCHORS * sizeof(uint16_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].start_idx, MAX_GRID * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].read_end_idx, MAX_GRID * sizeof(size_t));
        hipHostMalloc((void**)&host_mem_ptrs[i].cut_start_idx, MAX_GRID * sizeof(size_t));
        // set up GPU memory pointers
        hipMalloc(&device_mem_ptrs[i].d_ax, MAX_NUM_ANCHORS * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_ay, MAX_NUM_ANCHORS * sizeof(int64_t));
        hipMalloc(&device_mem_ptrs[i].d_range, MAX_NUM_ANCHORS * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_f, MAX_NUM_ANCHORS * sizeof(int32_t));
        hipMalloc(&device_mem_ptrs[i].d_p, MAX_NUM_ANCHORS * sizeof(uint16_t));    
        hipMalloc(&device_mem_ptrs[i].d_cut, MAX_NUM_CUT * sizeof(size_t));
        hipMalloc(&device_mem_ptrs[i].d_start_idx, sizeof(size_t)*MAX_GRID);
        hipMalloc(&device_mem_ptrs[i].d_read_end_idx, sizeof(size_t)*MAX_GRID);
        hipMalloc(&device_mem_ptrs[i].d_cut_start_idx, sizeof(size_t)*MAX_GRID);
        cudaCheck();
    }

    int count = total_iter / INPUT_BATCH_SIZE;
    input_iter* curr_arr = input_arr;
    
    // launch streams
    for (int i = 0; i < count; ++i) {
        // fprintf(stderr, "[M::%s] normal round i: %d\n", __func__, i);
        int iter = i;
        if (i >= NUM_STREAMS) {
            // int sync_iter = (i-NUM_STREAMS) % NUM_STREAMS;
            iter = -1;
            while (iter == -1) {
                for (int t = 0; t < NUM_STREAMS; ++t) {
                    if (!hipEventQuery(events[t])) {
                        iter = t;
                        hipEventDestroy(events[iter]);
                        hipEventCreate(&events[iter]);
                        break;
                    }
                }
            }

            // NOTE: streams fill up, sync previous stream
            // fprintf(stderr, "[M::%s] sync stream sync_iter: %d\n", __func__, sync_iter);
            // hipStreamSynchronize(streams[iter]);
            cudaCheck();
#ifdef DEBUG_CHECK
            int sync_iter = iter;
            // TODO: check correctness
            int index = host_mem_ptrs[sync_iter].index;
            // fprintf(stderr, "[M::%s] correctness check index: %d\n", __func__, index);
            input_iter *prev_arr = input_arr + index*INPUT_BATCH_SIZE;
            // range check
            int32_t* range = (int32_t*)malloc(sizeof(int32_t) * device_mem_ptrs[sync_iter].total_n);
            hipMemcpy(range, device_mem_ptrs[sync_iter].d_range, sizeof(int32_t)*device_mem_ptrs[sync_iter].total_n, hipMemcpyDeviceToHost);
            int64_t read_start = 0;
            // fprintf(stderr, "[M::%s] Start range check\n", __func__);
            for (int i = 0; i < INPUT_BATCH_SIZE; i++){
                debug_print_successor_range(range+read_start, prev_arr[i].n);
                debug_check_range(range + read_start, prev_arr[i].range, prev_arr[i].n);
                read_start += prev_arr[i].n;
            }
            free(range);
            // score check
            // fprintf(stderr, "[M::%s] Start score check\n", __func__);
            read_start = 0;
            for (int i = 0; i < INPUT_BATCH_SIZE; i++) {
                int64_t* p = (int64_t*)malloc(sizeof(int64_t)*prev_arr[i].n);
                p_rel2idx(host_mem_ptrs[sync_iter].p + read_start, p, prev_arr[i].n);
                debug_check_score(p, host_mem_ptrs[sync_iter].f + read_start, prev_arr[i].p, prev_arr[i].f, prev_arr[i].n);
                read_start += prev_arr[i].n;
                free(p);
            }
            // fprintf(stderr, "[M::%s] End score check\n", __func__);
#endif
            // TODO: read p, f
            // host_mem_ptrs[sync_iter].f
            // host_mem_ptrs[sync_iter].p
        }

        
        host_mem_ptrs[iter].index = i; // record batch index
        stream_range_selection(curr_arr, host_mem_ptrs + iter, device_mem_ptrs + iter, INPUT_BATCH_SIZE, (void *)(&streams[iter]));
        cudaCheck();
        stream_score_generation(curr_arr, host_mem_ptrs + iter, device_mem_ptrs + iter, INPUT_BATCH_SIZE, (void *)(&streams[iter]), (void *)(&events[iter]));
        cudaCheck();
        curr_arr += INPUT_BATCH_SIZE;

    }

    // launch extra batch shorter than INPUT_BATCH_SIZE
    if (count*INPUT_BATCH_SIZE < total_iter) {
        // fprintf(stderr, "[M::%s] extra round count: %d\n", __func__, count);
        int iter = count;
        if (count >= NUM_STREAMS) {
            // int sync_iter = (count-NUM_STREAMS) % NUM_STREAMS;
            iter = -1;
            while (iter == -1) {
                for (int t = 0; t < NUM_STREAMS; ++t) {
                    if (!hipEventQuery(events[t])) {
                        iter = t;
                        hipEventDestroy(events[iter]);
                        hipEventCreate(&events[iter]);
                        break;
                    }
                }
            }
            // NOTE: streams fill up, sync previous stream
            // fprintf(stderr, "[M::%s] sync stream sync_iter: %d\n", __func__, sync_iter);
            // hipStreamSynchronize(streams[iter]);
            cudaCheck();
#ifdef DEBUG_CHECK
            int sync_iter = iter;
            // TODO: check correctness
            int index = host_mem_ptrs[sync_iter].index;
            // fprintf(stderr, "[M::%s] correctness check index: %d\n", __func__, index);
            input_iter *prev_arr = input_arr + index*INPUT_BATCH_SIZE;
            // range check
            int32_t* range = (int32_t*)malloc(sizeof(int32_t) * device_mem_ptrs[sync_iter].total_n);
            hipMemcpy(range, device_mem_ptrs[sync_iter].d_range, sizeof(int32_t)*device_mem_ptrs[sync_iter].total_n, hipMemcpyDeviceToHost);
            int64_t read_start = 0;
            // fprintf(stderr, "[M::%s] Start range check\n", __func__);
            for (int i = 0; i < INPUT_BATCH_SIZE; i++){
                debug_print_successor_range(range+read_start, prev_arr[i].n);
                debug_check_range(range + read_start, prev_arr[i].range, prev_arr[i].n);
                read_start += prev_arr[i].n;
            }
            free(range);
            // score check
            read_start = 0;
            // fprintf(stderr, "[M::%s] Start score check\n", __func__);
            for (int i = 0; i < INPUT_BATCH_SIZE; i++) {
                int64_t* p = (int64_t*)malloc(sizeof(int64_t)*prev_arr[i].n);
                p_rel2idx(host_mem_ptrs[sync_iter].p + read_start, p, prev_arr[i].n);
                debug_check_score(p, host_mem_ptrs[sync_iter].f + read_start, prev_arr[i].p, prev_arr[i].f, prev_arr[i].n);
                read_start += prev_arr[i].n;
                free(p);
            }
            // fprintf(stderr, "[M::%s] End score check\n", __func__);
#endif
            // TODO: read p, f
            // host_mem_ptrs[sync_iter].f
            // host_mem_ptrs[sync_iter].p
        
        }
        // run the last kernel sets
        int size = total_iter - count*INPUT_BATCH_SIZE;
        host_mem_ptrs[iter].index = count; // record batch index
        stream_range_selection(curr_arr, host_mem_ptrs + iter, device_mem_ptrs + iter, size, (void *)(&streams[iter]));
        cudaCheck();
        stream_score_generation(curr_arr, host_mem_ptrs + iter, device_mem_ptrs + iter, size, (void *)(&streams[iter]), (void *)(&events[iter]));
        cudaCheck();
    }

    // sync all the streams
    for (int sync_iter = 0; sync_iter < NUM_STREAMS; ++sync_iter) {
        int index = host_mem_ptrs[sync_iter].index;
        // fprintf(stderr, "[M::%s] final sync all index: %d, sync_iter: %d\n", __func__, index, sync_iter);
        if (index >= 0) {
            hipStreamSynchronize(streams[sync_iter]);
            cudaCheck();
            // TODO: check correctness
#ifdef DEBUG_CHECK
            int batch_size = total_iter >= (index+1)*INPUT_BATCH_SIZE ? INPUT_BATCH_SIZE : (index+1)*INPUT_BATCH_SIZE-total_iter;
            // fprintf(stderr, "[M::%s] correctness check index: %d, batch_size: %d\n", __func__, index, batch_size);
            input_iter *prev_arr = input_arr + index*INPUT_BATCH_SIZE;
            // range check
            int32_t* range = (int32_t*)malloc(sizeof(int32_t) * device_mem_ptrs[sync_iter].total_n);
            hipMemcpy(range, device_mem_ptrs[sync_iter].d_range, sizeof(int32_t)*device_mem_ptrs[sync_iter].total_n, hipMemcpyDeviceToHost);
            int64_t read_start = 0;
            // fprintf(stderr, "[M::%s] Start range check\n", __func__);
            for (int i = 0; i < batch_size; i++){
                debug_print_successor_range(range+read_start, prev_arr[i].n);
                debug_check_range(range + read_start, prev_arr[i].range, prev_arr[i].n);
                read_start += prev_arr[i].n;
            }
            free(range);
            // score check
            read_start = 0;
            // fprintf(stderr, "[M::%s] Start score check\n", __func__);
            for (int i = 0; i < batch_size; i++) {
                int64_t* p = (int64_t*)malloc(sizeof(int64_t)*prev_arr[i].n);
                p_rel2idx(host_mem_ptrs[sync_iter].p + read_start, p, prev_arr[i].n);
                debug_check_score(p, host_mem_ptrs[sync_iter].f + read_start, prev_arr[i].p, prev_arr[i].f, prev_arr[i].n);
                read_start += prev_arr[i].n;
                free(p);
            }
            // fprintf(stderr, "[M::%s] End score check\n", __func__);
#endif
        } else break;
    }

    // free all memory
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(events[i]);
        cudaCheck();

        // free host memory
        hipHostFree(host_mem_ptrs[i].ax);
        hipHostFree(host_mem_ptrs[i].ay);
        hipHostFree(host_mem_ptrs[i].f);
        hipHostFree(host_mem_ptrs[i].p);
        hipHostFree(host_mem_ptrs[i].start_idx);
        hipHostFree(host_mem_ptrs[i].read_end_idx);
        hipHostFree(host_mem_ptrs[i].cut_start_idx);
        // free device memory
        hipFree(device_mem_ptrs[i].d_ax);
        hipFree(device_mem_ptrs[i].d_ay);
        hipFree(device_mem_ptrs[i].d_range);
        hipFree(device_mem_ptrs[i].d_f);
        hipFree(device_mem_ptrs[i].d_p);
        hipFree(device_mem_ptrs[i].d_cut);
        hipFree(device_mem_ptrs[i].d_start_idx);
        hipFree(device_mem_ptrs[i].d_read_end_idx);
        hipFree(device_mem_ptrs[i].d_cut_start_idx);
        cudaCheck();
    }
    delete[] streams;
    delete[] events;

    clk_end = time(NULL);
    // clk_end = clock();
    printf("[Stream chaining] ======Chaining Run Time: %lf secs\n", (double) (clk_end - clk_start));
    // printf("[Stream chaining] ======Chaining Run Time: %lf secs\n", ((double) (clk_end - clk_start)) / CLOCKS_PER_SEC);
    return;
}

/*********************** GPU loops end ************************/

